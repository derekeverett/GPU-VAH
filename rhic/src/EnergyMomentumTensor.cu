#include "hip/hip_runtime.h"
/*
 * EnergyMomentumTensor.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */
#include <math.h> // for math functions

#include "../include/EnergyMomentumTensor.cuh"
#include "../include/DynamicalVariables.cuh"
#include "../include/LatticeParameters.h"
#include "../include/CudaConfiguration.cuh"
#include "../include/EquationOfState.cuh"

#define MAX_ITERS 1000000000
#define VBAR 0.563624
#define EPS 0.1
//const PRECISION ACC = 1e-2;

__host__ __device__
PRECISION getTransverseFluidVelocityMagnitude(const FLUID_VELOCITY * const __restrict__ u, int s) {
		PRECISION u1 = u->ux[s];
		PRECISION u2 = u->uy[s];
		return sqrt(fabs(u1*u1+u2*u2));
}

__host__ __device__
int transverseFluidVelocityFromConservedVariables(PRECISION t, PRECISION ePrev, PRECISION uT_0,
PRECISION MB0, PRECISION MBT, PRECISION MB3, PRECISION PL, PRECISION Pi, double Ft, double x, double *uT,
int i, int jj, int k, double xi, double yj, double zk,
int fullTimeStepInversion
) {
	PRECISION uT0 = uT_0;	// initial guess for uT

	// Constants
	double Ft2 = Ft*Ft;
	double bT = x*MBT;
	double bL = x*x*MB0-Ft2*PL;
	double b = x*x+Ft2;

	double f,fp,DF;

	for(int j = 0; j < MAX_ITERS; ++j) {
		double e = MB0 - t*Ft*MB3 - uT0/sqrt(1 + uT0*uT0)*x*MBT;
		if(e < 0.0) return -1;
		double p = equilibriumPressure(e);
		double PtHat = 0.5*(e-PL);
		double Pt = PtHat + 1.5*Pi;

		double deduT = -x*MBT/pow(1 + uT0*uT0,1.5);
		double dPtduT = 0.5*deduT;

		f = uT0/sqrt(1+uT0*uT0)*(bL+b*Pt) - bT;
		fp = 1/pow(1 + uT0*uT0,1.5)*(bL+b*Pt)+uT0/sqrt(1+uT0*uT0)*b*dPtduT;

		if(fabs(fp)==0.0) fp = 1.e-16;

		DF = f/fp;

		*uT = uT0 - DF;

		if(isnan(*uT) || isinf(*uT) || *uT < 0 || *uT > 9.0072e+15) return -1;

		double DUT = fabs(*uT-uT0);
		double UT = fabs(*uT);
		if(DUT <=  1.e-7 * UT) return 0;
		uT0 = *uT;
	}
	return -1;
}

__host__ __device__
PRECISION energyDensityFromConservedVariables(PRECISION ePrev, PRECISION M0, PRECISION M, PRECISION Pi) {
#ifndef CONFORMAL_EOS
	PRECISION e0 = ePrev;	// initial guess for energy density
	for(int j = 0; j < MAX_ITERS; ++j) {
		PRECISION p = equilibriumPressure(e0);
		PRECISION cs2 = speedOfSoundSquared(e0);
		PRECISION cst2 = p/e0;

		PRECISION A = fmaf(M0,1-cst2,Pi);
		PRECISION B = fmaf(M0,M0+Pi,-M);
		PRECISION H = sqrtf(fabsf(A*A+4*cst2*B));
		PRECISION D = (A-H)/(2*cst2);

		PRECISION f = e0 + D;
		PRECISION fp = 1 - ((cs2 - cst2)*(B + D*H - ((cs2 - cst2)*cst2*D*M0)/e0))/(cst2*e0*H);

		PRECISION e = e0 - f/fp;
		if(fabsf(e - e0) <=  0.001 * fabsf(e)) return e;
		e0 = e;
	}
//	printf("Maximum number of iterations exceeded.\n");
	printf("Maximum number of iterations exceeded.\tePrev=%.3f,\tM0=%.3f,\t M=%.3f,\t Pi=%.3f\n",ePrev,M0,M,Pi);
	return e0;
#else
	return fabsf(sqrtf(fabsf(4 * M0 * M0 - 3 * M)) - M0);
#endif
}

__host__ __device__
void getInferredVariables(PRECISION t, const PRECISION * const __restrict__ q, PRECISION ePrev,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
PRECISION * const __restrict__ ut, PRECISION * const __restrict__ ux, PRECISION * const __restrict__ uy, PRECISION * const __restrict__ un,
double xi, double yj, double zk,
int fullTimeStepInversion
) {
	PRECISION ttt = q[0];
	PRECISION ttx = q[1];
	PRECISION tty = q[2];
	PRECISION ttn = q[3];
	PRECISION pl  = q[4];
#ifdef PIMUNU
	PRECISION pitt = q[5];
	PRECISION pitx = q[6];
	PRECISION pity = q[7];
	PRECISION pitn = q[8];
#else
	PRECISION pitt = 0;
	PRECISION pitx = 0;
	PRECISION pity = 0;
	PRECISION pitn = 0;
#endif
#ifdef W_TZ_MU
	PRECISION WtTz = q[15];
	PRECISION WxTz = q[16];
	PRECISION WyTz = q[17];
	PRECISION WnTz = q[18];
#else
	PRECISION WtTz = 0;
	PRECISION WxTz = 0;
	PRECISION WyTz = 0;
	PRECISION WnTz = 0;
#endif
	// \Pi
#ifdef PI
	PRECISION Pi = q[NUMBER_CONSERVED_VARIABLES-1];
#else
	PRECISION Pi = 0;
#endif

PRECISION M0 = ttt-pitt;
PRECISION M1 = ttx-pitx;
PRECISION M2 = tty-pity;
PRECISION M3 = ttn-pitn;

double t2 = t*t;

double M0PL = M0+pl;
if(M0PL==0.0) M0PL=1.e-16;

PRECISION A = M3/M0PL;
PRECISION At = t*A;
double B = WtTz/M0PL/t;
double Bt = t*B;
double At2 = At*At;
double Bt2 = Bt*Bt;
double F = (A-fabs(B)*sqrt(fabs(1-At2+Bt2)))/(1+Bt2);
double Ft = t*F;
double Ft2 = Ft*Ft;
PRECISION x = sqrt(fabs(1.-Ft2));

double MB0 = M0-2*WtTz*Ft/x;
double MB1 = M1-WxTz*Ft/x;
double MB2 = M2-WyTz*Ft/x;
double MB3 = M3-(1+Ft2)*WtTz/t/x;

double MBT = sqrt(MB1*MB1+MB2*MB2);
if(MBT==0.0) MBT=1.e-16;

double uT;
int status = -1;

status = transverseFluidVelocityFromConservedVariables(t, ePrev, uT_0, MB0, MBT, MB3, pl, Pi, Ft, x, &uT, i, j, k, xi, yj, zk, fullTimeStepInversion);

if(status == 0) {
	double C2 = 1.0+pow(uT,2.);
	double C = sqrt(C2);
	double U = uT/C;

	*ux=uT*MB1/MBT;
	*uy=uT*MB2/MBT;
	*un = F*C/x;
	*ut = C/x;
	//*ut = sqrt(C2+t2*pow((*un),2.));

	*e = MB0 - t*Ft*MB3 - U*x*MBT;
	*p = equilibriumPressure(*e);
}	else {
	*e = ePrev*.999;
	*p = equilibriumPressure(*e);
	*ux=0.0;
	*uy=0.0;
//		*un = F/x;
//		*ut = sqrt(1.0+t2*pow((*un),2.));
	*un = 0.0;
	*ut = 1.0;
}
//	if(*e > 1.1*ePrev && ePrev <= 0.1) {
//		*e = ePrev*.999;
//		*p = equilibriumPressure(*e);
//	}

if (isnan(*e) || isnan(*ut) || isnan(*ux) || isnan(*uy) || isnan(*un)) {
	printf("=======================================================================================\n");
	printf("found NaN in getInferredVariables.\n");
	printf("Grid point = (%d, %d, %d) = (%.3f, %.3f, %.3f)\n", i, j, k, xi, yj, zk);
	if(fullTimeStepInversion==0) printf("From semiDiscreteKurganovTadmorAlgorithm.\n");
	printf("t=%.3f\n",t);
	printf("uT=%.9f\n",uT);
	printf("ePrev=%.9f\n",ePrev);
	printf("A=%.9f;B=%.9f;F=%.9f;x=%.9f;\n",A,B,F,x);
	printf("e=%.9f;p=%.9f;\n",*e,*p);
	printf("ut=%.9f;ux=%.9f;uy=%.9f;un=%.9f;\n",*ut,*ux,*uy,*un);
	printf("MB1=%.9f\n",MB1);
	printf("MB2=%.9f\n",MB2);
	printf("MB0=%.3f,\t MBT=%.3f,\t MB3=%.3f,\tPL=%.3f,ePrev=%.3f\t,uT_0=%.3f\n", MB0, MBT, MB3,pl,ePrev,uT_0);
	printf("=======================================================================================\n");
	exit(-1);
}

return status;
}

__global__
void setInferredVariablesKernel(const CONSERVED_VARIABLES * const __restrict__ q,
	PRECISION * const __restrict__ e, PRECISION * const __restrict__ p, FLUID_VELOCITY * const __restrict__ u,
	PRECISION t
) {
	unsigned int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadID < d_nElements) {
		unsigned int k = threadID / (d_nx * d_ny) + N_GHOST_CELLS_M;
		unsigned int j = (threadID % (d_nx * d_ny)) / d_nx + N_GHOST_CELLS_M;
		unsigned int i = threadID % d_nx + N_GHOST_CELLS_M;
		unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);

		PRECISION q_s[NUMBER_CONSERVED_VARIABLES];
		q_s[0] = q->ttt[s];
		q_s[1] = q->ttx[s];
		q_s[2] = q->tty[s];
		q_s[3] = q->ttn[s];
		q_s[4] = q->pl[s];
		#ifdef PIMUNU
		q_s[5] = q->pitt[s];
		q_s[6] = q->pitx[s];
		q_s[7] = q->pity[s];
		q_s[8] = q->pitn[s];
		/****************************************************************************\
		q_s[8] = q->pixx[s];
		q_s[9] = q->pixy[s];
		q_s[10] = q->pixn[s];
		q_s[11] = q->piyy[s];
		q_s[12] = q->piyn[s];
		q_s[13] = q->pinn[s];
		/****************************************************************************/
		#endif

		#ifdef W_TZ_MU
		q_s[15] = q->WtTz[s];
		q_s[16] = q->WxTz[s];
		q_s[17] = q->WyTz[s];
		q_s[18] = q->WnTz[s];
		#endif

		#ifdef PI
		q_s[NUMBER_CONSERVED_VARIABLES-1] = q->Pi[s];
		#endif
		PRECISION uT = getTransverseFluidVelocityMagnitude(up, s);

		int status = getInferredVariables(t,q_s,e[s],uT,&_e,&_p,&ut,&ux,&uy,&un,i,j,k,x,y,z,1);
		if (status == 0) fTSolution[s] = 0.0;
		else fTSolution[s] = 1.0;

		e[s] = _e;
		p[s] = _p;
		u->ut[s] = ut;
		u->ux[s] = ux;
		u->uy[s] = uy;
		u->un[s] = un;
	}
}

//===================================================================
// Components of T^{\mu\nu} in (\tau,x,y,\eta_s)-coordinates
//===================================================================
__host__ __device__
PRECISION Ttt(PRECISION e, PRECISION p, PRECISION ut, PRECISION pitt) {
	return (e+p)*ut*ut-p+pitt;
}

__host__ __device__
PRECISION Ttx(PRECISION e, PRECISION p, PRECISION ut, PRECISION ux, PRECISION pitx) {
	return (e+p)*ut*ux+pitx;
}

__host__ __device__
PRECISION Tty(PRECISION e, PRECISION p, PRECISION ut, PRECISION uy, PRECISION pity) {
	return (e+p)*ut*uy+pity;
}

__host__ __device__
PRECISION Ttn(PRECISION e, PRECISION p, PRECISION ut, PRECISION un, PRECISION pitn) {
	return (e+p)*ut*un+pitn;
}

__host__ __device__
PRECISION Txx(PRECISION e, PRECISION p, PRECISION ux, PRECISION pixx) {
	return (e+p)*ux*ux+p+pixx;
}

__host__ __device__
PRECISION Txy(PRECISION e, PRECISION p, PRECISION ux, PRECISION uy, PRECISION pixy) {
	return (e+p)*ux*uy+pixy;
}

__host__ __device__
PRECISION Txn(PRECISION e, PRECISION p, PRECISION ux, PRECISION un, PRECISION pixn) {
	return (e+p)*ux*un+pixn;
}

__host__ __device__
PRECISION Tyy(PRECISION e, PRECISION p, PRECISION uy, PRECISION piyy) {
	return (e+p)*uy*uy+p+piyy;
}

__host__ __device__
PRECISION Tyn(PRECISION e, PRECISION p, PRECISION uy, PRECISION un, PRECISION piyn) {
	return (e+p)*uy*un+piyn;
}

__host__ __device__
PRECISION Tnn(PRECISION e, PRECISION p, PRECISION un, PRECISION pinn, PRECISION t) {
	return (e+p)*un*un+p/t/t+pinn;
}
